
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

__global__ void square(float * d_out, float * d_in){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 2048;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	square<<<2, 1024>>>(d_out, d_in);

	hipError_t err;
    	if ( hipSuccess != (err = hipGetLastError()) ){
        	fprintf(stderr, "CUDA error: %s\n", hipGetErrorString( err ) );
        	exit(-2);
    	}
	
	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
