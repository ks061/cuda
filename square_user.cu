
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

static const int NUM_BLOCKS = 64;
static const int NUM_THREADS = 1024; // per block	
static const int MAX_ARRAY_SIZE = NUM_BLOCKS * NUM_THREADS;

__global__ void square(double * d_out, int start){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double f = ((double)start) + ((double)idx);
    d_out[idx] = f * f;
}

void print_err() {
	printf("Usage: ./square_user [number of squares] [optional flags: -q]\n\n");
	exit(0);
}


void exec_kernel(int start, int end, int N, bool quiet) {	
	int ARRAY_BYTES = (end - start) * sizeof(double);
	double * d_out;
	double * h_out;
	
	// allocate GPU memory
	hipMalloc((void**) &d_out, ARRAY_BYTES);
	h_out = (double *)malloc(ARRAY_BYTES);

	// launch the kernel
	square<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, start);

	hipError_t err;
	if ( hipSuccess != (err = hipGetLastError()) ){
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString( err ) );
		exit(-2);
	}

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i = start; i < end; i++) {
		if (!quiet) {
			printf("%lf\n\n", h_out[i - start]);
		}
	//	if (i>510000) {printf("%d\n", i);}
		if (quiet && (i >= (N-4))) {
			printf("%lf\n\n", h_out[i - start]);
		}
	}

	hipFree(d_out);
}

int main(int argc, char ** argv) { 
	bool quiet = false;

	printf("\n");
	if (argc < 2) { 
		print_err(); 
	}
	if (argv[1] == NULL) { 
		print_err(); 
	} 
	if (argc > 2) {
		for (int i = 2; i < argc; i++) {
			if (!strcmp(argv[i], "-q")) {
				quiet = true;
			}
		}
	}

	int N = atoi(argv[1]);
	int curr_N;
	
	curr_N = N % MAX_ARRAY_SIZE;
	if (curr_N != 0) {
		exec_kernel(0, curr_N, N, quiet);
	}
	for (; curr_N < N; curr_N = curr_N + MAX_ARRAY_SIZE) {
		exec_kernel(curr_N, curr_N + MAX_ARRAY_SIZE, N, quiet);
	}

	return 0;
}
