
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <iostream>
#include <fstream>

static const int NUM_BLOCKS = 64;
static const int NUM_THREADS = 1024; // per block	
static const int MAX_ARRAY_SIZE = NUM_BLOCKS * NUM_THREADS;

__global__ void square(double * d_out, int start){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double f = (double) (start + idx);
    d_out[idx] = f * f;
}

void print_err() {
	printf("Usage: ./square_user [number of squares] [optional flags: -q]\n\n");
	exit(0);
}

void exec_kernel(int start, int end, int N, bool quiet) {	
	int ARRAY_BYTES = (end - start) * sizeof(double);
	double * d_out;
	double * h_out;
	
	// allocate GPU memory
	hipMalloc((void**) &d_out, ARRAY_BYTES);
	h_out = (double *)malloc(ARRAY_BYTES);

	// launch the kernel
	square<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, start);

	hipError_t err;
	if ( hipSuccess != (err = hipGetLastError()) ){
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString( err ) );
		exit(-2);
	}

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	/*
	for (int i = start; i < end; i++) {
		if (!quiet) {
			printf("%lf\n\n", h_out[i - start]);
		}
	//	if (i>510000) {printf("%d\n", i);}
		if (quiet && (i >= (N-4))) {
			printf("%lf\n\n", h_out[i - start]);
		}
	}
	*/

	hipFree(d_out);
	free(h_out);
}

int num_digits(int num) {
	int num_digits = 0;
	while (num != 0) {
		num = num / 10;
		num_digits = num_digits + 1;
	}
	return num_digits;
}

int main(int argc, char ** argv) { 
	bool quiet = false;

	printf("\n");
	if (argc < 2) { 
		print_err(); 
	}
	if (argv[1] == NULL) { 
		print_err(); 
	} 
	if (argc > 2) {
		for (int i = 2; i < argc; i++) {
			if (!strcmp(argv[i], "-q")) {
				quiet = true;
			}
		}
	}

	int MAX_N = atoi(argv[1]);
	
	std::ofstream out_file;
	out_file.open("square_user.csv");

	clock_t start;
	clock_t end;
	int curr_N;
	int cpu_time_ms;
	char * str1;
	char * str2;

	for (int N = 0; N < MAX_N; N = N + (MAX_N / 100)) { 
		start = clock();
		
		curr_N = N % MAX_ARRAY_SIZE;
		if (curr_N != 0) {
			exec_kernel(0, curr_N, N, quiet);
		}
		for (; curr_N < N; curr_N = curr_N + MAX_ARRAY_SIZE) {
			exec_kernel(curr_N, curr_N + MAX_ARRAY_SIZE, N, quiet);
		}

		end = clock();
		cpu_time_ms = (int)( ((double)(end - start)) / CLOCKS_PER_SEC * 1000 ); // * 1000 for s --> ms 

		str1 = (char *) malloc(num_digits(N) * sizeof(char));
		str2 = (char *) malloc(num_digits(cpu_time_ms) * sizeof(char));
		sprintf(str1, "%d", N);
		sprintf(str2, "%d", cpu_time_ms);
		out_file << str1;
		out_file << ",";
		out_file <<  str2;
		out_file << "\n";
		free(str1);
		free(str2);
	}
	out_file.close();	
	return 0;
}
