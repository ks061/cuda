
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <iostream>
#include <fstream>

static const int NUM_BLOCKS = 64;
static const int NUM_THREADS = 1024; // per block	
static const int MAX_ARRAY_SIZE = NUM_BLOCKS * NUM_THREADS;

void square(double * h_out, int start, int end){
	for (int i = start; i < end; i++) {
		h_out[i - start] = ((double)i) * ((double)i);
	}
}

void print_err() {
	printf("Usage: ./square_user [number of squares] [optional flags: -q]\n\n");
	exit(0);
}

void exec_square(int start, int end, int N, bool quiet) {	
	int ARRAY_BYTES = (end - start) * sizeof(double);
	double * h_out;
	h_out = (double *)malloc(ARRAY_BYTES);

	square(h_out, start, end);

	// print out the resulting array
	/*
	for (int i = start; i < end; i++) {
		if (!quiet) {
			printf("%lf\n\n", h_out[i - start]);
		}
	//	if (i>510000) {printf("%d\n", i);}
		if (quiet && (i >= (N-4))) {
			printf("%lf\n\n", h_out[i - start]);
		}
	}
	*/

	free(h_out);
}

int num_digits(int num) {
	int num_digits = 0;
	while (num != 0) {
		num = num / 10;
		num_digits = num_digits + 1;
	}
	return num_digits;
}

int main(int argc, char ** argv) { 
	bool quiet = false;

	printf("\n");
	if (argc < 2) { 
		print_err(); 
	}
	if (argv[1] == NULL) { 
		print_err(); 
	} 
	if (argc > 2) {
		for (int i = 2; i < argc; i++) {
			if (!strcmp(argv[i], "-q")) {
				quiet = true;
			}
		}
	}

	int MAX_N = atoi(argv[1]);

	std::ofstream out_file;
	out_file.open("square_user_host.csv");
	
	clock_t start;
	clock_t end;
	int curr_N;
	int cpu_time_ms;
	char * str1;
	char * str2;
	
	for (int N = 0; N < MAX_N; N = N + (MAX_N / 100)) {
		start = clock();

		curr_N = N % MAX_ARRAY_SIZE;
		if (curr_N != 0) {
			exec_square(0, curr_N, N, quiet);
		}
		for (; curr_N < N; curr_N = curr_N + MAX_ARRAY_SIZE) {
			exec_square(curr_N, curr_N + MAX_ARRAY_SIZE, N, quiet);
		}

		end = clock();
		cpu_time_ms = (int)( ((double)(end-start)) / CLOCKS_PER_SEC * 1000 ); // * 1000 for s --> ms

		str1 = (char *) malloc(num_digits(N) * sizeof(char));
                str2 = (char *) malloc(num_digits(cpu_time_ms) * sizeof(char));
                sprintf(str1, "%d", N);
                sprintf(str2, "%d", cpu_time_ms);
                out_file << str1;
                out_file << ",";
                out_file <<  str2;
                out_file << "\n";
                free(str1);
                free(str2);		
	}
	out_file.close();
	return 0;
}
